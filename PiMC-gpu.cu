
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void init(unsigned int seed, hiprandState_t* states, int n) {

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
  hiprand_init(seed,
              i,
              0,
              &states[i]);
    }
}

// kernel to calculate random numbers
__global__ void randoms (hiprandState_t* states, float* numbers, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        numbers[i] = (hiprand_uniform(&states[i]));
    }

}
__global__ void calc_pi(float* rand_nums, float* sum, int n){

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        atomicAdd(sum, sqrt(1 - rand_nums[i]*rand_nums[i]));
    }
    
}

int main(int argc, char* argv[]){

    int N; 

// check for the appropriate number of command line arguments
    if (argc < 2){
        printf("Too few arguments\n"); 
        exit(1); 
    }

    N = atoi(argv[1]); 
    dim3 dimBlock(1024);
    dim3 dimGrid((int)ceil((float)N / 1024)); 

// check if the command line argument is a negative number. 
    if(N < 0){
        printf("No such thing as Negative Iterations !\n");
        exit(1);
    }

// keep track of seed value for every thread
  hiprandState_t* states;

  // allocate space on GPU for random states
  hipMalloc((void**) &states, N*sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<dimGrid, dimBlock>>>(time(0), states, N);
  hipDeviceSynchronize();

  // allocate array of unsigned ints on CPU and GPU
  float nums[N];
  float* dev_nums;
  hipMalloc((void**) &dev_nums, N*sizeof(float));

  // obtain a uniformly random distriubtion of integers, maximum N
  randoms<<<dimGrid, dimBlock>>>(states, dev_nums, N);
  hipDeviceSynchronize();

  // copy random distribution of integers back to host
  hipMemcpy(nums, dev_nums, N*sizeof(float), hipMemcpyDeviceToHost);
  

  // allocate for sum
  float sum = 0; 
  float* dev_sum; 
  hipMalloc((void**) &dev_sum, sizeof(float)); 

  // copy the initial value to the gpu
  hipMemcpy(dev_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

  // calling the kernel the first time
  calc_pi<<<dimGrid,dimBlock>>>(dev_nums, dev_sum, N);
  hipDeviceSynchronize();

  // reset sum and give it back to device
  sum =0; 
  hipMemcpy(dev_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

  // time to start timing
  hipEvent_t start;
  hipEventCreate(&start);
  hipEvent_t stop;
  hipEventCreate(&stop);

  // start timer
  hipEventRecord(start,0);
  // call kernel second time to get the accurate timing
  calc_pi<<<dimGrid,dimBlock>>>(dev_nums, dev_sum, N);
  hipDeviceSynchronize();

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  float diff;
  hipEventElapsedTime(&diff, start, stop);
  printf("time: %f ms\n", diff);

  // deallocate timers
  hipEventDestroy(start);
  hipEventDestroy(stop);

  

  // copy minimum value to host
  hipMemcpy(&sum, dev_sum, sizeof(float), hipMemcpyDeviceToHost);

  float pi = sum/N *4 ; 
   printf("Pi: %f\n", pi); 

  hipFree(states);
  hipFree(dev_nums);
  hipFree(dev_sum);

}
